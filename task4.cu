#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>


// функция изменения матрицы уравнения теплопроводности
__global__ void calculate(double *CudaArr, double *CudaNewArr, size_t Matrix)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x; //вычисления линейного индекса элемента внутри сетки 
    size_t j =  blockDim.y * blockIdx.y + threadIdx.y; 
    int index = i * Matrix + j;
    if (!(i == 0 || j == 0))
        CudaNewArr[index] = 0.25 * (CudaArr[(i - 1) * Matrix + j] + CudaArr[(i + 1) * Matrix + j] + CudaArr[index - 1] + CudaArr[index + 1]);
}


// функция разницы матриц
__global__ void subtraction(double* CudaArr, double* CudaNewArr, size_t Matrix)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;  
    size_t j =  blockDim.y * blockIdx.y + threadIdx.y;
    int idx = i * Matrix + j; 
    if ((i < Matrix && j < Matrix))
	    CudaNewArr[idx] = CudaArr[idx] - CudaNewArr[idx];
}

// функция востановления границ матрицы
__global__ void restore(double* arr, int size){
	size_t i = threadIdx.x;
	arr[i] = 10.0 + i * 10.0 / (size - 1);
	arr[i * size] = 10.0 + i * 10.0 / (size - 1);
	arr[size - 1 + i * size] = 20.0 + i * 10.0 / (size - 1);
	arr[size * (size - 1) + i] = 20.0 + i * 10.0 / (size - 1);
}

int find_threads(int size){
	if (size%32==0)
		return size/1024;

	return int(size/1024)+1;

}

int main(int argc, char* argv[]) {
    
    double time_spent = 0.0;

    clock_t begin = clock(); 

    // Convert command line arguments to integers
    int Matrix = atoi(argv[1]);
    double accuracy = atof(argv[2]);
    int iterations = atoi(argv[3]);

    hipSetDevice(1);
    
    // создание потока
    hipStream_t stream;
    hipStreamCreate(&stream);

    // создание графа
    hipGraph_t graph;
    hipGraphExec_t graph_exec;

    // выделяем память на gpu через cuda для 2 сеток
    double *CudaArr, *CudaNewArr;
    hipMalloc((void **)&CudaArr, sizeof(double) * Matrix * Matrix);
    hipMalloc((void **)&CudaNewArr, sizeof(double) * Matrix * Matrix);

    restore<<<1, Matrix>>>(CudaArr, Matrix);
    hipMemcpy(CudaNewArr, CudaArr, sizeof(double) * Matrix * Matrix, hipMemcpyHostToDevice);

    // выделяем память на gpu. Хранение ошибки на device
    double *max_err = 0;
    hipMalloc((void **)&max_err, sizeof(double));

    size_t tempStorageBytes = 0;
    double *tempStorage = NULL;

    // получаем размер временного буфера для редукции
    hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, CudaNewArr, max_err, Matrix * Matrix, stream);

    // выделяем память для буфера
    hipMalloc(&tempStorage, tempStorageBytes);

    dim3 t(32,32); //определяю количество нитей в каждом блоке 
    int b = find_threads(Matrix);

    ///////////////////////////////////////////////////////////////создаем граф
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    // Compute new values
    for (size_t i = 0; i < 100; i += 2) {
        //количество блоков, потоков, разделяемая память, поток
        calculate<<<b, t, 0, stream>>>(CudaArr, CudaNewArr, Matrix);
        calculate<<<b, t, 0, stream>>>(CudaNewArr, CudaArr, Matrix);
    }
    subtraction<<<b, t, 0, stream>>>(CudaArr, CudaNewArr, Matrix);

    // Compute maximum error using CUB
    hipcub::DeviceReduce::Max(tempStorage, tempStorageBytes, CudaNewArr, max_err, Matrix * Matrix, stream);
    restore<<<1, Matrix, 0, stream>>>(CudaNewArr, Matrix);

    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0);

    /////////////////////////////////////////////////////////////////

    // Main loop
    double err = 1;
    int iter = 0;

    while (err > accuracy && iter < iterations) {

        hipGraphLaunch(graph_exec, stream);
        // запись ошибки в переменную
        hipMemcpy(&err, max_err, sizeof(double), hipMemcpyDeviceToHost);
        iter+=100;  
    }

    printf("Final result: %d, %0.6lf\n", iter, err);

    // удаление потока и графа
    hipStreamDestroy(stream);
    hipGraphDestroy(graph);

    hipFree(CudaArr);
    hipFree(CudaNewArr);

    clock_t end = clock();
    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Time elapsed: %f\n", time_spent);

    return 0;
}
